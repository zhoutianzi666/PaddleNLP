#include "hip/hip_runtime.h"
// Copyright (c) 2025 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "helper.h"
#include "paddle/extension.h"

template <int THREADBLOCK_SIZE, bool EAGLE>
__global__ void draft_model_preprocess_kernel(
    int64_t* draft_tokens,
    int64_t* input_ids,
    bool* stop_flags,
    int* seq_lens_this_time,
    int* seq_lens_encoder,
    int* seq_lens_decoder,
    int64_t* step_idx,
    int* first_token_record,
    bool* not_need_stop,
    const int64_t* accept_tokens,
    const int* accept_num,
    const int* base_model_seq_lens_encoder,
    const int* base_model_seq_lens_decoder,
    const int64_t* base_model_step_idx,
    const bool* base_model_stop_flags,
    int64_t* base_model_draft_tokens,
    const int bsz,
    const int max_draft_token,
    const int accept_tokens_len,
    const int draft_tokens_len,
    const int input_ids_len,
    const int base_model_draft_tokens_len) {
  typedef hipcub::BlockReduce<int64_t, THREADBLOCK_SIZE> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int64_t not_stop_flag = 0;

  int tid = threadIdx.x;

  if (tid < bsz) {
    auto base_model_step_idx_now = base_model_step_idx[tid];
    auto* accept_tokens_now = accept_tokens + tid * accept_tokens_len;
    auto* draft_tokens_now = draft_tokens + tid * draft_tokens_len;
    auto accept_num_now = accept_num[tid];
    auto* input_ids_now = input_ids + tid * input_ids_len;
    auto* base_model_draft_tokens_now =
        base_model_draft_tokens + tid * base_model_draft_tokens_len;
#pragma unroll
    for (int i = 1; i < base_model_draft_tokens_len; i++) {
      base_model_draft_tokens_now[i] = -1;
    }

    if (!base_model_stop_flags[tid]) {
      not_stop_flag = 1;
      // 1. first token
      if (base_model_step_idx_now == 0) {
        seq_lens_this_time[tid] = 0;
        not_stop_flag = 0;
      } else if (base_model_step_idx_now == 1 && first_token_record[tid] > 0) {
        // Can be extended to first few tokens
        seq_lens_encoder[tid] = first_token_record[tid];
        first_token_record[tid] = -1;
        stop_flags[tid] = false;
        int64_t base_model_first_token = accept_tokens_now[0];
        int position = base_model_seq_lens_decoder[tid];
        if (EAGLE) {
          input_ids_now[position - 1] = base_model_first_token;
          seq_lens_this_time[tid] = base_model_seq_lens_decoder[tid];
        } else {
          input_ids_now[position] = base_model_first_token;
          seq_lens_this_time[tid] = base_model_seq_lens_decoder[tid] + 1;
        }
      } else if (accept_num_now <=
                 max_draft_token) /*Accept partial draft tokens*/ {
        // Base Model reject stop
        if (stop_flags[tid]) {
          stop_flags[tid] = false;
          seq_lens_decoder[tid] = base_model_seq_lens_decoder[tid];
          step_idx[tid] = base_model_step_idx[tid];
        } else {
          seq_lens_decoder[tid] -= max_draft_token - accept_num_now;
          step_idx[tid] -= max_draft_token - accept_num_now;
        }
        int64_t modified_token = accept_tokens_now[accept_num_now - 1];
        draft_tokens_now[0] = modified_token;
        seq_lens_this_time[tid] = 1;

      } else /*Accept all draft tokens*/ {
        draft_tokens_now[1] = accept_tokens_now[max_draft_token];
        seq_lens_this_time[tid] = 2;
      }
    } else {
      stop_flags[tid] = true;
      seq_lens_this_time[tid] = 0;
      seq_lens_decoder[tid] = 0;
    }
  }
  __syncthreads();
  int64_t not_stop_flag_sum = BlockReduce(temp_storage).Sum(not_stop_flag);
  if (tid == 0) {
    not_need_stop[0] = not_stop_flag_sum > 0;
  }
}


void DraftModelPreprocess(const paddle::Tensor& draft_tokens,
                          const paddle::Tensor& input_ids,
                          const paddle::Tensor& stop_flags,
                          const paddle::Tensor& seq_lens_this_time,
                          const paddle::Tensor& seq_lens_encoder,
                          const paddle::Tensor& seq_lens_decoder,
                          const paddle::Tensor& step_idx,
                          const paddle::Tensor& first_token_record,
                          const paddle::Tensor& not_need_stop,
                          const paddle::Tensor& accept_tokens,
                          const paddle::Tensor& accept_num,
                          const paddle::Tensor& base_model_seq_lens_encoder,
                          const paddle::Tensor& base_model_seq_lens_decoder,
                          const paddle::Tensor& base_model_step_idx,
                          const paddle::Tensor& base_model_stop_flags,
                          const paddle::Tensor& base_model_draft_tokens,
                          const int max_draft_token,
                          const std::string& draft_type) {
  int real_bsz = seq_lens_this_time.shape()[0];
  int accept_tokens_len = accept_tokens.shape()[1];
  int input_ids_len = input_ids.shape()[1];
  int draft_tokens_len = draft_tokens.shape()[1];
  auto cu_stream = seq_lens_this_time.stream();
  constexpr int BlockSize = 256;
  int base_model_draft_tokens_len = base_model_draft_tokens.shape()[1];
  auto not_need_stop_gpu =
      not_need_stop.copy_to(seq_lens_this_time.place(), false);


  if (draft_type == "eagle") {
    draft_model_preprocess_kernel<BlockSize, true>
        <<<1, BlockSize, 0, cu_stream>>>(
            const_cast<int64_t*>(draft_tokens.data<int64_t>()),
            const_cast<int64_t*>(input_ids.data<int64_t>()),
            const_cast<bool*>(stop_flags.data<bool>()),
            const_cast<int*>(seq_lens_this_time.data<int>()),
            const_cast<int*>(seq_lens_encoder.data<int>()),
            const_cast<int*>(seq_lens_decoder.data<int>()),
            const_cast<int64_t*>(step_idx.data<int64_t>()),
            const_cast<int*>(first_token_record.data<int>()),
            const_cast<bool*>(not_need_stop_gpu.data<bool>()),
            accept_tokens.data<int64_t>(),
            accept_num.data<int>(),
            base_model_seq_lens_encoder.data<int>(),
            base_model_seq_lens_decoder.data<int>(),
            base_model_step_idx.data<int64_t>(),
            base_model_stop_flags.data<bool>(),
            const_cast<int64_t*>(base_model_draft_tokens.data<int64_t>()),
            real_bsz,
            max_draft_token,
            accept_tokens_len,
            draft_tokens_len,
            input_ids_len,
            base_model_draft_tokens_len);
  } else {
    draft_model_preprocess_kernel<BlockSize, false>
        <<<1, BlockSize, 0, cu_stream>>>(
            const_cast<int64_t*>(draft_tokens.data<int64_t>()),
            const_cast<int64_t*>(input_ids.data<int64_t>()),
            const_cast<bool*>(stop_flags.data<bool>()),
            const_cast<int*>(seq_lens_this_time.data<int>()),
            const_cast<int*>(seq_lens_encoder.data<int>()),
            const_cast<int*>(seq_lens_decoder.data<int>()),
            const_cast<int64_t*>(step_idx.data<int64_t>()),
            const_cast<int*>(first_token_record.data<int>()),
            const_cast<bool*>(not_need_stop_gpu.data<bool>()),
            accept_tokens.data<int64_t>(),
            accept_num.data<int>(),
            base_model_seq_lens_encoder.data<int>(),
            base_model_seq_lens_decoder.data<int>(),
            base_model_step_idx.data<int64_t>(),
            base_model_stop_flags.data<bool>(),
            const_cast<int64_t*>(base_model_draft_tokens.data<int64_t>()),
            real_bsz,
            max_draft_token,
            accept_tokens_len,
            draft_tokens_len,
            input_ids_len,
            base_model_draft_tokens_len);
  }


  auto not_need_stop_cpu =
      not_need_stop_gpu.copy_to(not_need_stop.place(), false);
  bool* not_need_stop_data = const_cast<bool*>(not_need_stop.data<bool>());
  not_need_stop_data[0] = not_need_stop_cpu.data<bool>()[0];
}


PD_BUILD_OP(draft_model_preprocess)
    .Inputs({"draft_tokens",
             "input_ids",
             "stop_flags",
             "seq_lens_this_time",
             "seq_lens_encoder",
             "seq_lens_decoder",
             "step_idx",
             "first_token_record",
             "not_need_stop",
             "accept_tokens",
             "accept_num",
             "base_model_seq_lens_encoder",
             "base_model_seq_lens_decoder",
             "base_model_step_idx",
             "base_model_stop_flags",
             "base_model_draft_tokens"})
    .Outputs({"draft_tokens_out",
              "input_ids_out",
              "stop_flags_out",
              "seq_lens_this_time_out",
              "seq_lens_encoder_out",
              "seq_lens_decoder_out",
              "step_idx_out",
              "not_need_stop_out",
              "first_token_record_out"})
    .Attrs({"max_draft_token: int", "draft_type: std::string"})
    .SetInplaceMap({{"draft_tokens", "draft_tokens_out"},
                    {"input_ids", "input_ids_out"},
                    {"stop_flags", "stop_flags_out"},
                    {"seq_lens_this_time", "seq_lens_this_time_out"},
                    {"seq_lens_encoder", "seq_lens_encoder_out"},
                    {"seq_lens_decoder", "seq_lens_decoder_out"},
                    {"step_idx", "step_idx_out"},
                    {"not_need_stop", "not_need_stop_out"},
                    {"first_token_record", "first_token_record_out"}})
    .SetKernelFn(PD_KERNEL(DraftModelPreprocess));